/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), hipGetErrorName(result), func);
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

namespace cg = cooperative_groups;

#define THREADS_PER_BLOCK 512
#define GRAPH_LAUNCH_ITERATIONS 3

typedef struct callBackData {
  const char *fn_name;
  double *data;
} callBackData_t;

void CUDART_CB myHostNodeCallback(void *data) {
  // Check status of GPU after stream operations are done
  callBackData_t *tmp = (callBackData_t *)(data);
  // checkCudaErrors(tmp->status);

  double *result = (double *)(tmp->data);
  char *function = (char *)(tmp->fn_name);
  printf("[%s] Host callback final reduced sum = %lf\n", function, *result);
}

void cudaGraphsManual() {
  hipStream_t streamForGraph;
  hipGraph_t graph;
  std::vector<hipGraphNode_t> nodeDependencies;
  double result_h = 1.0;

  checkCudaErrors(hipStreamCreate(&streamForGraph));
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipGraphNode_t hostNode;
  hipHostNodeParams hostParams = {0};
  hostParams.fn = myHostNodeCallback;
  callBackData_t hostFnData;
  hostFnData.data = &result_h;
  hostFnData.fn_name = "cudaGraphsManual";
  hostParams.userData = &hostFnData;

  checkCudaErrors(hipGraphAddHostNode(&hostNode, graph,
                                       nodeDependencies.data(),
                                       nodeDependencies.size(), &hostParams));

  hipGraphNode_t *nodes = NULL;
  size_t numNodes = 0;
  checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
  printf("\nNum of nodes in the graph created manually = %zu\n", numNodes);

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

  hipGraph_t clonedGraph;
  hipGraphExec_t clonedGraphExec;
  checkCudaErrors(hipGraphClone(&clonedGraph, graph));
  checkCudaErrors(
      hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0));

  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
  }

  checkCudaErrors(hipStreamSynchronize(streamForGraph));

  printf("Cloned Graph Output.. \n");
  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(clonedGraphExec, streamForGraph));
  }
  checkCudaErrors(hipStreamSynchronize(streamForGraph));

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphExecDestroy(clonedGraphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipGraphDestroy(clonedGraph));
  checkCudaErrors(hipStreamDestroy(streamForGraph));
}

int main(int argc, char **argv) {
  printf("threads per block  = %d\n", THREADS_PER_BLOCK);
  printf("Graph Launch iterations = %d\n", GRAPH_LAUNCH_ITERATIONS);

  cudaGraphsManual();

  return EXIT_SUCCESS;
}

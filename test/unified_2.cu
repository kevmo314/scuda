// #include <iostream>
// #include <math.h>

// struct Operation {
//     float *x;
//     float *y;
//     int n;
// };

// // CUDA kernel to add elements of two arrays
// __global__ void add(Operation *op) {
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     int stride = blockDim.x * gridDim.x;

//     printf("The X is: %x\n", op->x[0]);
//     printf("The Y is: %x\n", op->y[0]);
//     for (int i = index; i < op->n; i += stride)
//     {
//       op->y[i] = op->x[i] + op->y[i];
//       printf("The value is: %f\n", op->y[i]);
//     }
// }

// int main(void) {
//     Operation host_op; // Host structure
//     Operation *device_op; // Device structure

//     // Initialize array size
//     host_op.n = 100;

//     // Allocate memory for device operation struct
//     cudaMalloc(&device_op, sizeof(Operation));

//     // Allocate memory for x and y arrays on the device
//     cudaMalloc(&host_op.x, host_op.n * sizeof(float));
//     cudaMalloc(&host_op.y, host_op.n * sizeof(float));

//     // Initialize x and y arrays on the host
//     float *host_x = new float[host_op.n];
//     float *host_y = new float[host_op.n];
//     for (int i = 0; i < host_op.n; i++) {
//         host_x[i] = 1.0f;
//         host_y[i] = 2.0f;
//     }

//     // Copy x and y arrays from host to device
//     cudaMemcpy(host_op.x, host_x, host_op.n * sizeof(float), cudaMemcpyHostToDevice);
//     cudaMemcpy(host_op.y, host_y, host_op.n * sizeof(float), cudaMemcpyHostToDevice);

//     std::cout << "BEFORE COPY DEVICE :" << &host_op.x << std::endl;
//     std::cout << "BEFORE COPY DEVICE :" << &host_op.y << std::endl;

//     // Copy host operation struct to device
//     cudaMemcpy(device_op, &host_op, sizeof(Operation), cudaMemcpyHostToDevice);

//     std::cout << "AFTER POINTER DEVICE :" << &device_op << std::endl;
//     std::cout << "AFTER POINTER HOST :" << &host_op << std::endl;
//     std::cout << "AFTER COPY DEVICE :" << &device_op->x << std::endl;
//     std::cout << "AFTER COPY DEVICE :" << &device_op->y << std::endl;

//     // Launch kernel
//     int blockSize = 256;
//     int numBlocks = (host_op.n + blockSize - 1) / blockSize;
//     add<<<numBlocks, blockSize>>>(device_op);

//     // Wait for GPU to finish before accessing results
//     cudaDeviceSynchronize();

//     // Copy results from device to host
//     cudaMemcpy(host_y, host_op.y, host_op.n * sizeof(float), cudaMemcpyDeviceToHost);

//     // Log results for debugging
//     std::cout << "Results (y = x + y):" << std::endl;
//     for (int i = 0; i < host_op.n; i++) {
//         std::cout << "y[" << i << "] = " << host_y[i] << " (expected: 3.0)" << std::endl;
//     }

//     // Check for errors (all values should be 3.0f)
//     float maxError = 0.0f;
//     for (int i = 0; i < host_op.n; i++) {
//         maxError = fmax(maxError, fabs(host_y[i] - 3.0f));
//     }

//     // Free device memory
//     cudaFree(host_op.x);
//     cudaFree(host_op.y);
//     cudaFree(device_op);

//     // Free host memory
//     delete[] host_x;
//     delete[] host_y;

//     return 0;
// }



// // ******UNIFIED MEMORY EXAMPLE BELOW*******



#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

struct Operation {
    float *x;
    float *y;
    int n;
};

// CUDA kernel to add elements of two arrays
__global__ void add(Operation *op) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    printf("The X is: %x\n", op->x[0]);
    printf("The Y is: %x\n", op->y[0]);
    for (int i = index; i < op->n; i += stride)
    {
      op->y[i] = op->x[i] + op->y[i];
      printf("The value is: %f\n", op->y[i]);
    }
}

int main(void) {
    Operation *op;

    // Allocate Unified Memory -- accessible from CPU or GPU
    hipMallocManaged(&op, sizeof(Operation));
    op->n = 100;

    hipMallocManaged(&op->x, op->n * sizeof(float));
    hipMallocManaged(&op->y, op->n * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < op->n; i++) {
        op->x[i] = 1.0f;
        op->y[i] = 2.0f;
    }

    // Launch kernel on n elements on the GPU
    int blockSize = 256;
    int numBlocks = (op->n + blockSize - 1) / blockSize;

    std::cout << "numBlocks: " << numBlocks << std::endl;
    std::cout << "N: " << op->n << std::endl;

    add<<<numBlocks, blockSize>>>(op);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Log results for debugging
    std::cout << "Results (y = x + y):" << std::endl;
    for (int i = 0; i < op->n; i++) {
        std::cout << "y[" << i << "] = " << op->y[i] << " (expected: 3.0)" << std::endl;
    }

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < op->n; i++) {
        maxError = fmax(maxError, fabs(op->y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(op->x);
    hipFree(op->y);
    hipFree(op);

    return 0;
}

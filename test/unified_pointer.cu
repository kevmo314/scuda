
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

struct Operation {
    float *x;
    float *y;
    int n;
};

// CUDA kernel to add elements of two arrays
__global__ void add(Operation *op) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    printf("The X is: %x\n", op->x[0]);
    printf("The Y is: %x\n", op->y[0]);
    for (int i = index; i < op->n; i += stride)
    {
      op->y[i] = op->x[i] + op->y[i];
    }
}

int main(void) {
    Operation *op;

    // Allocate Unified Memory -- accessible from CPU or GPU
    hipMallocManaged(&op, sizeof(Operation));
    op->n = 100;

    hipMallocManaged(&op->x, op->n * sizeof(float));
    hipMallocManaged(&op->y, op->n * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < op->n; i++) {
        op->x[i] = 1.0f;
        op->y[i] = 2.0f;
    }

    // Launch kernel on n elements on the GPU
    int blockSize = 256;
    int numBlocks = (op->n + blockSize - 1) / blockSize;

    std::cout << "numBlocks: " << numBlocks << std::endl;
    std::cout << "X: " << &op->x << std::endl;

    add<<<numBlocks, blockSize>>>(op);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Log results for debugging
    std::cout << "Results (y = x + y):" << std::endl;
    for (int i = 0; i < op->n; i++) {
        std::cout << "y[" << i << "] = " << op->y[i] << " (expected: 3.0)" << std::endl;
    }

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < op->n; i++) {
        maxError = fmax(maxError, fabs(op->y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(op->x);
    hipFree(op->y);
    hipFree(op);

    return 0;
}

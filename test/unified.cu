#include <hip/hip_runtime.h>
#include <iostream>

// CUDA Kernel to add elements of two arrays
__global__ void addKernel(int *a, int *b, int *c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] * b[idx];
    }
}

int main() {
    // Define array size
    const int size = 10;
    const int bytes = size * sizeof(int);

    std::cout << "HELLO" << std::endl;

    // Unified memory allocation
    int *a, *b, *c;
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    // Initialize arrays on the CPU
    for (int i = 0; i < size; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Define kernel launch parameters
    const int threadsPerBlock = 256;
    const int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    std::cout << "launching kernel..." << std::endl;

    std::cout << "pointer a: " << a << std::endl;
    std::cout << "pointer b: " << b << std::endl;
    std::cout << "pointer c: " << c << std::endl;

    // Launch the kernel
    addKernel<<<blocks, threadsPerBlock>>>(a, b, c, size);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Display results
    std::cout << "Results:\n";
    for (int i = 0; i < size; ++i) {
        std::cout << "a[" << i << "] + b[" << i << "] = " << c[i] << "\n";
    }

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}

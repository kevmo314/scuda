#include <hip/hip_runtime.h>
#include <iostream>

// CUDA Kernel to add elements of two arrays
// __global__ void addKernel(int *a, int *b, int *c, int size) {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     if (idx < size) {
//         c[idx] = a[idx] * b[idx];
//     }
// }

__global__ void mulKernel(int *a, int *c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] * 100;
    }
}

int main() {

    // Define array size
    const int size = 10;
    const int bytes = size * sizeof(int);

    // Unified memory allocation
    int *a, *c;

    hipMallocManaged(&c, bytes);
    hipMallocManaged(&a, bytes);

    // Initialize arrays on the CPU
    for (int i = 0; i < size; ++i) {
        a[i] = i;
    }

    // Define kernel launch parameters
    const int threadsPerBlock = 256;
    const int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    std::cout << "launching kernel..." << std::endl;

    for (int i = 0; i < size; ++i) {
        std::cout << "a[" << i << "] + b[" << i << "] = " << a[i] << "\n";
    }

    // Launch the kernel
    mulKernel<<<blocks, threadsPerBlock>>>(a, c, size);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Display results
    std::cout << "Results:\n";
    for (int i = 0; i < size; ++i) {
        std::cout << "a[" << i << "] + b[" << i << "] = " << c[i] << "\n";
    }

    // Free unified memory
    hipFree(a);
    hipFree(c);

    return 0;
}

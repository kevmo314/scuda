#include <iostream>
#include <hip/hip_runtime.h>
#include <cudnn.h>

/**
 * Minimal example to apply sigmoid activation on a tensor 
 * using cuDNN with Unified Memory.
 **/
int main(int argc, char** argv)
{
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    std::cout << "Found " << numGPUs << " GPUs." << std::endl;
    hipSetDevice(0); // use GPU0
    int device;
    struct hipDeviceProp_t devProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&devProp, device);
    std::cout << "Compute capability:" << devProp.major << "." << devProp.minor << std::endl;

    cudnnHandle_t handle_;
    cudnnCreate(&handle_);
    std::cout << "Created cuDNN handle" << std::endl;

    // Create the tensor descriptor
    cudnnDataType_t dtype = CUDNN_DATA_FLOAT;
    cudnnTensorFormat_t format = CUDNN_TENSOR_NCHW;

    int n = 1, c = 1, h = 1, w = 10;
    int NUM_ELEMENTS = n * c * h * w;
    cudnnTensorDescriptor_t x_desc;

    cudnnCreateTensorDescriptor(&x_desc);
    cudnnSetTensor4dDescriptor(x_desc, format, dtype, n, c, h, w);

    // Allocate unified memory for the tensor
    float *x;
    hipMallocManaged(&x, NUM_ELEMENTS * sizeof(float));

    // Initialize the tensor
    for (int i = 0; i < NUM_ELEMENTS; i++) x[i] = i * 1.00f;

    std::cout << "Original array: ";
    for (int i = 0; i < NUM_ELEMENTS; i++) std::cout << x[i] << " ";
    std::cout << std::endl;

    // Synchronize to ensure data is accessible on the device
    hipDeviceSynchronize();

    // Create activation function descriptor
    float alpha[1] = {1};
    float beta[1] = {0.0};
    cudnnActivationDescriptor_t sigmoid_activation;
    cudnnActivationMode_t mode = CUDNN_ACTIVATION_SIGMOID;
    cudnnNanPropagation_t prop = CUDNN_NOT_PROPAGATE_NAN;
    cudnnCreateActivationDescriptor(&sigmoid_activation);
    cudnnSetActivationDescriptor(sigmoid_activation, mode, prop, 0.0f);

    cudnnActivationForward(
        handle_,
        sigmoid_activation,
        alpha,
        x_desc,
        x,
        beta,
        x_desc,
        x
    );

    // Synchronize to ensure data is updated on the host
    hipDeviceSynchronize();

    cudnnDestroy(handle_);
    std::cout << "Destroyed cuDNN handle." << std::endl;

    std::cout << "New array: ";
    for (int i = 0; i < NUM_ELEMENTS; i++) std::cout << x[i] << " ";
    std::cout << std::endl;

    hipFree(x);

    return 0;
}

#include <arpa/inet.h>
#include <cstdlib>
#include <cstring>
#include <functional>
#include <future>
#include <iostream>
#include <memory>
#include <nvml.h>
#include <pthread.h>
#include <stdio.h>
#include <string>
#include <sys/socket.h>
#include <sys/uio.h>
#include <thread>
#include <unistd.h>
#include <unordered_map>

#include "codegen/gen_server.h"
#include "rpc.h"

#define DEFAULT_PORT 14833
#define MAX_CLIENTS 10

int request_handler(conn_t *conn) {
  unsigned int op;

  // Attempt to read the operation code from the client
  if (read(conn->connfd, &op, sizeof(unsigned int)) < 0)
    return -1;

  auto opHandler = get_handler(op);

  if (opHandler == NULL) {
    std::cerr << "Unknown or unsupported operation: " << op << std::endl;
    return -1;
  }

  return opHandler(conn);
}

void client_handler(int connfd) {
  conn_t conn = {connfd};
  if (pthread_mutex_init(&conn.read_mutex, NULL) < 0 ||
      pthread_mutex_init(&conn.write_mutex, NULL) < 0) {
    std::cerr << "Error initializing mutex." << std::endl;
    return;
  }

#ifdef VERBOSE
  printf("Client connected.\n");
#endif

  if (pthread_mutex_lock(&conn.read_mutex) < 0) {
    std::cerr << "Error locking mutex." << std::endl;
  }
  while (1) {
    while (conn.read_id != 0)
      pthread_cond_wait(&conn.read_cond, &conn.read_mutex);

    int n = read(connfd, &conn.read_id, sizeof(int));
    if (n == 0) {
      printf("client disconnected, loop continuing. \n");
      break;
    } else if (n < 0) {
      printf("error reading from client.\n");
      break;
    }

    if (conn.read_id < 0) {
      // this is a response to an existing request, notify everyone else
      // and spin again.
      if (pthread_cond_broadcast(&conn.read_cond) < 0) {
        std::cerr << "Error broadcasting condition or unlocking mutex."
                  << std::endl;
        break;
      }
      continue;
    } else {
      // TODO: this can't be multithreaded as some of the __cuda* functions
      // assume that they are running in the same thread as the one that
      // calls hipLaunchKernel. we'll need to find a better way to map
      // function calls to threads. maybe each rpc maps to an optional
      // thread id that is passed to the handler?
      if (request_handler(&conn) < 0)
        std::cerr << "Error handling request." << std::endl;
    }
  }

  if (pthread_mutex_destroy(&conn.read_mutex) < 0 ||
      pthread_mutex_destroy(&conn.write_mutex) < 0)
    std::cerr << "Error destroying mutex." << std::endl;
  close(connfd);
}

int main() {
  int port = DEFAULT_PORT;
  struct sockaddr_in servaddr, cli;
  int sockfd = socket(AF_INET, SOCK_STREAM, 0);
  if (sockfd == -1) {
    printf("Socket creation failed.\n");
    exit(EXIT_FAILURE);
  }

  char *p = getenv("SCUDA_PORT");

  if (p == NULL) {
    port = DEFAULT_PORT;
  } else {
    port = atoi(p);
  }

  // Bind the socket
  memset(&servaddr, 0, sizeof(servaddr));
  servaddr.sin_family = AF_INET;
  servaddr.sin_addr.s_addr = INADDR_ANY;
  servaddr.sin_port = htons(port);

  const int enable = 1;
  if (setsockopt(sockfd, SOL_SOCKET, SO_REUSEADDR, &enable, sizeof(int)) < 0) {
    printf("Socket bind failed.\n");
    exit(EXIT_FAILURE);
  }

  if (bind(sockfd, (struct sockaddr *)&servaddr, sizeof(servaddr)) != 0) {
    printf("Socket bind failed.\n");
    exit(EXIT_FAILURE);
  }

  if (listen(sockfd, MAX_CLIENTS) != 0) {
    printf("Listen failed.\n");
    exit(EXIT_FAILURE);
  }

  printf("Server listening on port %d...\n", port);

  // Server loop
  while (1) {
    socklen_t len = sizeof(cli);
    int connfd = accept(sockfd, (struct sockaddr *)&cli, &len);

    if (connfd < 0) {
      std::cerr << "Server accept failed." << std::endl;
      continue;
    }

    std::thread client_thread(client_handler, connfd);

    // detach the thread so it runs independently
    client_thread.detach();
  }

  close(sockfd);
  return 0;
}
